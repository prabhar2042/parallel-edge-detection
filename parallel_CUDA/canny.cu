#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "canny.h"

typedef uchar3 Pixel_cuda;

#define THREADS_PER_BLOCK 16

__constant__ float blur_kernel[3][3] = {{0.0625, 0.125, 0.0625},
                                        {0.125, 0.25, 0.125},
                                        {0.0625, 0.125, 0.0625}};

void flatten_pixel_array(Image img, Pixel_cuda *flat_pixels)
{

    for (int i = 0; i < img.height; i++)
    {
        for (int j = 0; j < img.width; j++)
        {
            flat_pixels[i * img.width + j].x = img.pixels[i][j].rgb.r;
            flat_pixels[i * img.width + j].y = img.pixels[i][j].rgb.g;
            flat_pixels[i * img.width + j].z = img.pixels[i][j].rgb.b;
        }
    }
}

void unflatten_pixel_array(Image img, unsigned char *flat_pixels)
{
    for (int i = 0; i < img.height; i++)
    {
        for (int j = 0; j < img.width; j++)
        {
            img.pixels[i][j].gray.value = static_cast<unsigned char>(flat_pixels[i * img.width + j]);
        }
    }
}

__global__ void rgb_to_gray_kernel(Pixel_cuda *pixels, unsigned char *output, int height, int width)
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= height || col >= width)
    {
        return;
    }

    uchar3 p = pixels[row * width + col];

    double grayValue = (0.299 * p.x) +
                       (0.587 * p.y) +
                       (0.114 * p.z);

    output[row * width + col] = static_cast<unsigned char>(grayValue);
    // output[row * width + col] = static_cast<unsigned char>(100);
}

__global__ void gaussianblur_kernel(unsigned char *gray, unsigned char *output, int height, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= height || col >= width)
    {
        return;
    }

    if (row < 1 || col < 1 || row >= (height - 1) || col >= (width - 1))
    {
        // Pixels at the image borders are left unchanged
        output[row * width + col] = gray[row * width + col];
        return;
    }

    float sum = 0.0;
    for (int i = -1; i <= 1; ++i)
    {
        for (int j = -1; j <= 1; ++j)
        {
            sum += blur_kernel[i + 1][j + 1] * gray[(row + i) * width + (col + j)];
        }
    }

    output[row * width + col] = static_cast<unsigned char>(sum);
}

/*
__global__ void sobel_kernel(Image *img)
{
}

__global__ void nms_kernel(Image *img)
{
}

__global__ void threshholding_kernel(Image *img)
{
}

__global__ void edge_tracking_kernel(Image *img){}

*/

void canny(exec_time &time, char *read_file, char *write_file)
{
    // Allocate memory for host image on stack
    Image host_img;

    // Read image
    read_PPM(host_img, read_file);

    // Flatten Pixel array
    Pixel_cuda *host_flat_pixels = (Pixel_cuda *)malloc(host_img.width * host_img.height * sizeof(Pixel_cuda));
    flatten_pixel_array(host_img, host_flat_pixels);

    // Allocate memory for RGB image on GPU
    Pixel_cuda *device_pixels;
    hipMalloc((void **)&device_pixels, host_img.width * host_img.height * sizeof(Pixel_cuda));

    // Allocate memory for grayscale image on GPU
    unsigned char *device_gray;
    hipMalloc((void **)&device_gray, host_img.width * host_img.height * sizeof(unsigned char));

    // Allocate memory for grayscale image output on GPU
    unsigned char *device_gray_out;
    hipMalloc((void **)&device_gray_out, host_img.width * host_img.height * sizeof(unsigned char));

    // Copy RGB image to GPU
    hipMemcpy(device_pixels, host_flat_pixels, host_img.width * host_img.height * sizeof(Pixel_cuda), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 blockDim(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 gridDim((host_img.width + blockDim.x - 1) / blockDim.x,
                 (host_img.height + blockDim.y - 1) / blockDim.y);

    double start, end;

    // Start image detection on GPU
    printf("pixel before grey: %d\n", host_flat_pixels[0].x);

    // 1. Convert image to grayscale
    start = clock();
    rgb_to_gray_kernel<<<gridDim, blockDim>>>(device_pixels, device_gray, host_img.height, host_img.width);
    hipDeviceSynchronize();
    end = clock();
    time.rgb_to_gray = ((double)end - (double)start) / CLOCKS_PER_SEC;
    time.total += time.rgb_to_gray;

    /*// 2. Gaussian Blur
    start = clock();
    gaussianblur_kernel<<<gridDim, blockDim>>>(device_gray, device_gray_out, host_img.height, host_img.width);
    hipDeviceSynchronize();
    end = clock();
    time.gaussian_blur = ((double)end - (double)start) / CLOCKS_PER_SEC;
    time.total += time.gaussian_blur;*/

    // Copy image data from device to host
    unsigned char *host_gray = (unsigned char *)malloc(host_img.width * host_img.height * sizeof(unsigned char));
    hipMemcpy(host_gray, device_gray, host_img.width * host_img.height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    /*for (int i = 0; i < (host_img.width * host_img.height); i++)
    {
        printf("pixel after grey: %d\n", host_gray[i]);
        if (i > 25)
        {
            break;
        }
    }*/

    // Unflatten pixel array
    unflatten_pixel_array(host_img, host_gray);

    // Write image
    write_PPM(host_img, write_file);

    // Free device memory space
    hipFree(device_pixels);
    hipFree(device_gray);

    // Free host memory
    free(host_flat_pixels);
    free(host_gray);
}
