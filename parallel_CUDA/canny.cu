#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "ppm.h"
#include "image.h"

/**
 * @brief reads rgb image
 *
 * @param img
 * @param filename
 * @return true
 * @return false
 */
bool read_PPM(Image &img, const char *filename)
{
    std::ifstream file(filename, std::ios::binary);

    if (!file)
    {
        std::cerr << "Failed to open image file." << std::endl;
        return FAIL;
    }

    // Read PPM header
    std::string magic;
    int maxval;
    file >> magic >> img.width >> img.height >> maxval;
    if (magic != "P3" && magic != "P6")
    {
        std::cerr << "Invalid PPM format." << std::endl;
        return FAIL;
    }

    // Allocate memory for image data
    img.pixels.resize(img.height, std::vector<Pixel>(img.width));

    // Read image data
    if (magic == "P3")
    {
        // P3 format: ASCII
        int r, g, b;
        for (int row = 0; row < img.height; ++row)
        {
            for (int col = 0; col < img.width; ++col)
            {
                file >> r >> g >> b;
                img.pixels[row][col].rgb.r = static_cast<unsigned char>(r);
                img.pixels[row][col].rgb.g = static_cast<unsigned char>(g);
                img.pixels[row][col].rgb.b = static_cast<unsigned char>(b);
            }
        }
    }
    else
    {
        // P6 format: binary
        if (maxval != 255)
        {
            std::cerr << "Unsupported maxval value." << std::endl;
            return FAIL;
        }
        file.get(); // Read and discard newline character after header
        for (int row = 0; row < img.height; ++row)
        {
            file.read(reinterpret_cast<char *>(&img.pixels[row][0]), img.width * sizeof(Pixel));
        }
    }
    file.close();

    return PASS;
}

/**
 * @brief writes gray scale image
 *
 * @param filename
 * @param img
 */
void write_PPM(const Image &img, const char *filename)
{
    std::ofstream file(filename, std::ios::binary);
    if (!file)
    {
        std::cerr << "Failed to create image file." << std::endl;
        return;
    }

    // Write PPM header
    file << "P5\n"
         << img.width << " " << img.height << "\n255\n";

    // Write image data
    for (int row = 0; row < img.height; ++row)
    {
        for (int col = 0; col < img.width; ++col)
        {
            const Pixel &pixel = img.pixels[row][col];
            unsigned char value = pixel.gray.value;
            file.write(reinterpret_cast<const char *>(&value), sizeof(unsigned char));
        }
    }
    file.close();
}


__global__ void rgb_to_gray_kernel(Image *img) {

}

__global__ void blur_kernel(Image *img) {

}

__global__ void sobel_kernel(Image *img) {

}

__global__ void nms_kernel(Image *img) {

}

__global__ void threshholding_kernel(Image *img) {

}

__global__ void edge_tracking_kernel(Image *img) {

}

void canny(char *read_file, char *write_file) {

    clock_t start, end;
    time.total = 0;
    Image img;

    // read image
    read_PPM(img, read_file);

    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    rgb_to_gray_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result);

    blur_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result);
    
    sobel_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result);

    nms_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result);

    threshholding_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result);

    edge_tracking_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result);

}
