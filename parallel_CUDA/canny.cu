#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <iostream>
#include <fstream>
using namespace std;

#include <cmath>
#include "canny.h"

#define THREADS_PER_BLOCK 32
#define PI_BY_8 = 0.785398163397448309616 / 2;

typedef uchar3 Pixel_cuda;
typedef float3 grad_cuda;

__device__ inline double magnitude(double dX, double dY)
{
    return sqrt(dX * dX + dY * dY);
}

__device__ inline double direction(double dX, double dY)
{
    return atan2(dY, dX); // range [-pi,pi] radians
}

double find_max_grad(grad_cuda *grad, int height, int width)
{
    double max_grad = 0;
    for (int row = 0; row < height; row++)
    {
        for (int col = 0; col < width; col++)
        {
            if (grad[row * width + col].x > max_grad)
            {
                max_grad = grad[row * width + col].x;
            }
        }
    }
    return max_grad;
}

__device__ inline bool check_ngh(grad_cuda *grad, int height, int width, int row, int col)
{

    // iterate through pixel neighborhood

    for (int i = -1; i <= 1; ++i)
    {
        for (int j = -1; j <= 1; ++j)
        {
            if (((row + i) >= 0) && (row + i) < height)
            {
                if (((col + j) >= 0) && (col + j) < width)
                {
                    if ((int) (grad[(row + i) * width + (col + j)].z) == strong)
                    {
                        return true;
                    }
                }
            }
        }
    }
    return false;
}

void flatten_pixel_array(Image img, Pixel_cuda *flat_pixels)
{

    for (int i = 0; i < img.height; i++)
    {
        for (int j = 0; j < img.width; j++)
        {
            flat_pixels[i * img.width + j].x = img.pixels[i][j].rgb.r;
            flat_pixels[i * img.width + j].y = img.pixels[i][j].rgb.g;
            flat_pixels[i * img.width + j].z = img.pixels[i][j].rgb.b;
        }
    }
}

void unflatten_pixel_array(Image img, unsigned char *flat_pixels)
{
    for (int i = 0; i < img.height; i++)
    {
        for (int j = 0; j < img.width; j++)
        {
            img.pixels[i][j].gray.value = static_cast<unsigned char>(flat_pixels[i * img.width + j]);
        }
    }
}

__global__ void rgb_to_gray_kernel(Pixel_cuda *pixels, unsigned char *output, int height, int width)
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= height || col >= width)
    {
        return;
    }

    uchar3 p = pixels[row * width + col];

    double grayValue = (0.299 * p.x) +
                       (0.587 * p.y) +
                       (0.114 * p.z);

    output[row * width + col] = static_cast<unsigned char>(grayValue);
    // output[row * width + col] = static_cast<unsigned char>(100);
}

__global__ void gaussianblur_kernel(unsigned char *gray, unsigned char *output, int height, int width)
{
    float blur_kernel[3][3] = {
        {0.0625, 0.125, 0.0625},
        {0.125, 0.25, 0.125},
        {0.0625, 0.125, 0.0625}};

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= height || col >= width)
    {
        return;
    }

    if (row < 1 || col < 1 || row >= (height - 1) || col >= (width - 1))
    {
        // Pixels at the image borders are left unchanged
        output[row * width + col] = gray[row * width + col];
        return;
    }

    float sum = 0.0;
    for (int i = -1; i <= 1; ++i)
    {
        for (int j = -1; j <= 1; ++j)
        {
            sum += blur_kernel[i + 1][j + 1] * gray[(row + i) * width + (col + j)];
        }
    }

    output[row * width + col] = static_cast<unsigned char>(sum);
}

__global__ void sobel_kernel(unsigned char *gray, grad_cuda *grads, unsigned char *gray_out, int height, int width)

{
    float sobel_kernelX[3][3] = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
    float sobel_kernelY[3][3] = {{-1, -2, -1}, {0, 0, 0}, {1, 2, 1}};

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= height || col >= width)
    {
        return;
    }

    if (row < 1 || col < 1 || row >= (height - 1) || col >= (width - 1))
    {
        // Pixels at the image borders are left unchanged
        gray_out[row * width + col] = gray[row * width + col];
        grads[row * width + col].x = static_cast<float>(gray[row * width + col]);
        return;
    }

    double dX = 0, dY = 0;

    for (int i = -1; i <= 1; ++i)
    {
        for (int j = -1; j <= 1; ++j)
        {
            dX += sobel_kernelX[i + 1][j + 1] * gray[(row + i) * width + (col + j)]; // for gradient along x
            dY += sobel_kernelY[i + 1][j + 1] * gray[(row + i) * width + (col + j)]; // for gradient along y
        }
    }

    // Calculate magnitude and direction
    double mag = magnitude(dX, dY);
    double dir = direction(dX, dY);
    
    // Store gradient information in grads array
    grads[row * width + col].x = mag;
    grads[row * width + col].y = dir;
    gray_out[row * width + col] = (unsigned char)mag;
}

__global__ void nms_kernel(unsigned char *gray, unsigned char *gray_out,
                           grad_cuda *grads, grad_cuda *grads_out,
                           int height, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= height || col >= width)
    {
        return;
    }

    int linear_idx = row * width + col;

    if (row < 1 || col < 1 || row >= (height - 1) || col >= (width - 1))
    {
        // Pixels at the image borders are left unchanged
        grads_out[linear_idx] = grads[linear_idx];
        gray_out[linear_idx] = gray[linear_idx];
        return;
    }

    double mag = grads[linear_idx].x;
    double dir = grads[linear_idx].y;
    double pi_by_8 = 0.785398163397448309616 / 2;

    // Determine adjacent pixels based on gradient direction
    double pixel1_mag, pixel2_mag;
    if ((dir > (-1 * pi_by_8) && dir <= pi_by_8) || (dir > (7 * pi_by_8) && dir <= (-7 * pi_by_8))) 
    {
        pixel1_mag = grads[row * width + col - 1].x;
        pixel2_mag = grads[row * width + col + 1].x;
    }
    else if ((dir > pi_by_8 && dir <= (3 * pi_by_8)) || (dir > (-7 * pi_by_8) && dir <= (-5 * pi_by_8)))
    {
        pixel1_mag = grads[(row - 1) * width + col + 1].x;
        pixel2_mag = grads[(row + 1) * width + col - 1].x;
    }
    else if ((dir > (3 * pi_by_8) && dir <= (5 * pi_by_8)) || (dir > (-5 * pi_by_8) && dir <= (-3 * pi_by_8)))
    {
        pixel1_mag = grads[(row - 1) * width + col].x;
        pixel2_mag = grads[(row + 1) * width + col].x;
    }
    else
    {
        pixel1_mag = grads[(row - 1) * width + col - 1].x;
        pixel2_mag = grads[(row + 1) * width + col + 1].x;
    }


    // Suppress non-maximum pixels
    if (mag <= pixel1_mag || mag <= pixel2_mag)
    {
        grads_out[linear_idx].x = 0.0;
        gray_out[linear_idx] = 0;
    }
}

__global__ void threshholding_kernel(unsigned char *gray, unsigned char *gray_out,
                                    grad_cuda *grads, grad_cuda *grads_out, double max_grad,
                                    int height, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= height || col >= width)
    {
        return;
    }

    const double high_threshold_ratio = 0.1;
    const double low_threshold_ratio = 0.05;

    double high_thres = max_grad * high_threshold_ratio;
    double low_thres = max_grad * low_threshold_ratio;

    int linear_idx = row * width + col;

    if (grads[linear_idx].x >= high_thres)
    {
        grads_out[linear_idx].z = static_cast<float>(strong);
    }
    else if ((grads[linear_idx].x >= low_thres) && (grads[linear_idx].x < high_thres))
    {
        grads_out[linear_idx].z = static_cast<float>(weak);
    }
    else
    {
        grads_out[linear_idx].z = static_cast<float>(no_edge);
        grads_out[linear_idx].x = 0;
        gray_out[linear_idx] = 0;
    }
}

__global__ void edge_tracking_kernel(unsigned char *gray, grad_cuda *grads, unsigned char *gray_out, int height, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= height || col >= width)
    {
        return;
    }

    int linear_idx = row * width + col;

    if ((int)(grads[linear_idx].z) == weak)
    {
        if (!check_ngh(grads, height, width, row, col))
        {
            // grads[linear_idx].x = 0;
            gray[linear_idx] = 0;
        }
    }
}

void canny(exec_time &time, char *read_file, char *write_file)
{
    // Allocate memory for host image on stack
    Image host_img;

    // Read image
    read_PPM(host_img, read_file);

    // Flatten Pixel array
    Pixel_cuda *host_flat_pixels = (Pixel_cuda *)malloc(host_img.width * host_img.height * sizeof(Pixel_cuda));
    flatten_pixel_array(host_img, host_flat_pixels);

    // Allocate memory for RGB image on GPU
    Pixel_cuda *device_pixels;
    hipMalloc((void **)&device_pixels, host_img.width * host_img.height * sizeof(Pixel_cuda));

    // Allocate memory for grayscale image on GPU
    unsigned char *device_gray;
    hipMalloc((void **)&device_gray, host_img.width * host_img.height * sizeof(unsigned char));

    // Allocate memory for grayscale image output on GPU
    unsigned char *device_gray_out;
    hipMalloc((void **)&device_gray_out, host_img.width * host_img.height * sizeof(unsigned char));

    // Allocate memory for image gradients on GPU
    grad_cuda *device_grads;
    hipMalloc((void **)&device_grads, host_img.width * host_img.height * sizeof(grad_cuda));

    // Allocate memory for image gradient ouputs on GPU
    grad_cuda *device_grads_out;
    hipMalloc((void **)&device_grads_out, host_img.width * host_img.height * sizeof(grad_cuda));

    // Copy RGB image to GPU
    hipMemcpy(device_pixels, host_flat_pixels, host_img.width * host_img.height * sizeof(Pixel_cuda), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 blockDim(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 gridDim((host_img.width + blockDim.x - 1) / blockDim.x,
                 (host_img.height + blockDim.y - 1) / blockDim.y);

    double start, end;

    // Start image detection on GPU
    printf("pixel before grey: %d\n", host_flat_pixels[0].x);

    // 1. Convert image to grayscale
    start = clock();
    rgb_to_gray_kernel<<<gridDim, blockDim>>>(device_pixels, device_gray, host_img.height, host_img.width);
    hipDeviceSynchronize();
    end = clock();
    time.rgb_to_gray = ((double)end - (double)start) / CLOCKS_PER_SEC;
    time.total += time.rgb_to_gray;

    // 2. Gaussian Blur
    start = clock();
    gaussianblur_kernel<<<gridDim, blockDim>>>(device_gray, device_gray_out, host_img.height, host_img.width);
    hipDeviceSynchronize();
    end = clock();
    time.gaussian_blur = ((double)end - (double)start) / CLOCKS_PER_SEC;
    time.total += time.gaussian_blur;


    // 3. Determine intensity gradient
    start = clock();
    sobel_kernel<<<gridDim, blockDim>>>(device_gray_out, device_grads, device_gray, host_img.height, host_img.width);
    hipDeviceSynchronize();
    end = clock();
    time.sobel = ((double)end - (double)start) / CLOCKS_PER_SEC;
    time.total += time.sobel;

    
    /* grad_cuda *temp;
    temp = (grad_cuda *)malloc(host_img.width * host_img.height * sizeof(grad_cuda));
    hipMemcpy(temp, device_grads, host_img.width * host_img.height * sizeof(grad_cuda), hipMemcpyDeviceToHost); */

    // 4. Non Maximum seperation
    start = clock();
    nms_kernel<<<gridDim, blockDim>>>(device_gray, device_gray_out, device_grads, device_grads_out, host_img.height, host_img.width);
    hipDeviceSynchronize();
    end = clock();
    time.nms = ((double)end - (double)start) / CLOCKS_PER_SEC;
    time.total += time.nms;

    // Find max grad
    grad_cuda *host_grads;
    host_grads = (grad_cuda *)malloc(host_img.width * host_img.height * sizeof(grad_cuda));
    hipMemcpy(host_grads, device_grads_out, host_img.width * host_img.height * sizeof(grad_cuda), hipMemcpyDeviceToHost);

    double max_grad = find_max_grad(host_grads, host_img.height, host_img.width);

    // 5. Double Thresholding
    start = clock();
    threshholding_kernel<<<gridDim, blockDim>>>(device_gray_out, device_gray,
                                                device_grads_out, device_grads,
                                                max_grad,
                                                host_img.height, host_img.width);
    hipDeviceSynchronize();
    end = clock();
    time.double_thres = ((double)end - (double)start) / CLOCKS_PER_SEC;
    time.total += time.double_thres;

    // 6. Edge Tracking
    start = clock();
    edge_tracking_kernel<<<gridDim, blockDim>>>(device_gray, device_grads,
                                                device_gray_out, host_img.height, host_img.width);
    hipDeviceSynchronize();
    end = clock();
    time.edge_track = ((double)end - (double)start) / CLOCKS_PER_SEC;
    time.total += time.edge_track;


    // Copy image data from device to host
    unsigned char *host_gray = (unsigned char *)malloc(host_img.width * host_img.height * sizeof(unsigned char));
    hipMemcpy(host_gray, device_gray_out, host_img.width * host_img.height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Unflatten pixel array
    unflatten_pixel_array(host_img, host_gray);

    // Write image
    write_PPM(host_img, write_file);

    // Free device memory space
    hipFree(device_pixels);
    hipFree(device_gray);
    hipFree(device_gray_out);
    hipFree(device_grads);
    hipFree(device_grads_out);

    // Free host memory
    free(host_flat_pixels);
    free(host_gray);
    free(host_grads);
}
