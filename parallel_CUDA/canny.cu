#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "canny.h"

#define THREADS_PER_BLOCK 16


__constant__ float blur_kernel[3][3] = {{0.0625, 0.125, 0.0625},
                                 {0.125, 0.25, 0.125},
                                 {0.0625, 0.125, 0.0625}};

__global__ void rgb_to_gray_kernel(Image device_img)
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= device_img.height || col >= device_img.width)
    {
        return;
    }

    double grayValue = 0.299 * device_img.pixels[row][col].rgb.r +
                       0.587 * device_img.pixels[row][col].rgb.g +
                       0.114 * device_img.pixels[row][col].rgb.b;

    device_img.pixels[row][col].gray.value = static_cast<unsigned char>(grayValue);
}

__global__ void gaussianblur_kernel(Image device_img, Image device_result)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= device_img.height || col >= device_img.width || row == 0 || col == 0)
    {
        return;
    }

    float sum = 0.0;
    for (int i = -1; i <= 1; ++i)
    {
        for (int j = -1; j <= 1; ++j)
        {
            sum += blur_kernel[i + 1][j + 1] * device_img.pixels[row + i][col + j].gray.value;
        }
    }
    // Store the blurred pixel in the new image
    device_result.pixels[row][col].gray.value = static_cast<unsigned char>(sum);
}
/*
__global__ void sobel_kernel(Image *img)
{
}

__global__ void nms_kernel(Image *img)
{
}

__global__ void threshholding_kernel(Image *img)
{
}

__global__ void edge_tracking_kernel(Image *img){}

*/

void canny(exec_time &time, char *read_file, char *write_file)
{
    // allocate memory for host image on stack
    Image host_img;

    printf("about to read ppm\n");

    // read image
    read_PPM(host_img, read_file);

    // allocate memory for image on GPU
    Image device_img;
    hipMalloc((void **)&device_img.pixels, host_img.width * host_img.height * sizeof(Pixel));
    hipMalloc((void **)&device_img.grads, host_img.width * host_img.height * sizeof(gradient));
    hipMalloc((void **)&device_img.width, sizeof(int));
    hipMalloc((void **)&device_img.height, sizeof(int));

    // copy image to GPU
    hipMemcpy(device_img.pixels, &(host_img.pixels[0][0]), host_img.width * host_img.height * sizeof(Pixel), hipMemcpyHostToDevice);
    hipMemcpy(&device_img.height, &(host_img.height), sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(&device_img.width, &(host_img.width), sizeof(int), hipMemcpyHostToDevice);


    
    // Define block and grid dimensions
    dim3 blockDim(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 gridDim((host_img.width + blockDim.x - 1) / blockDim.x,
                 (host_img.height + blockDim.y - 1) / blockDim.y);

    double start, end;

    // start image detection on GPU

    start = clock();
    rgb_to_gray_kernel<<<blockDim, gridDim>>>(device_img); // 1. convert image to grayscale
    hipDeviceSynchronize();
    end = clock();
    time.rgb_to_gray = ((double)end - (double)start) / CLOCKS_PER_SEC;
    time.total += time.rgb_to_gray;

    
    /*
        start = clock();
        gaussianblur_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result); // 2. Gaussian Blur
        hipDeviceSynchronize();
        end = clock();
        time.gaussian_blur = ((double)end - (double)start) / CLOCKS_PER_SEC;
        time.total += time.gaussian_blur;

        start = clock();
        sobel_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result); // 3. Determine intensity gradient
        hipDeviceSynchronize();
        end = clock();
        time.sobel = ((double)end - (double)start) / CLOCKS_PER_SEC;
        time.total += time.sobel;

        start = clock();
        nms_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result); // 4. Non Maximum seperation
        hipDeviceSynchronize();
        end = clock();
        time.nms = ((double)end - (double)start) / CLOCKS_PER_SEC;
        time.total += time.nms;

        start = clock();
        threshholding_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result); // 5. Double Thresholding
        hipDeviceSynchronize();
        end = clock();
        time.double_thres = ((double)end - (double)start) / CLOCKS_PER_SEC;
        time.total += time.double_thres;

        start = clock();
        edge_tracking_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result); // 6. Edge Tracking
        hipDeviceSynchronize();
        end = clock();
        time.edge_track = ((double)end - (double)start) / CLOCKS_PER_SEC;
        time.total += time.edge_track;

        */
    
    // Copy image data from device to host

    printf("I am here\n");
    hipMemcpy(&(host_img.pixels), &(device_img.pixels), host_img.width * host_img.height * sizeof(Pixel), hipMemcpyDeviceToHost);
    hipMemcpy(&(host_img.grads), &(device_img.grads), host_img.width * host_img.height * sizeof(gradient), hipMemcpyDeviceToHost);

    // write image
    write_PPM(host_img, write_file);

    // free device memory space
    hipFree(device_img.pixels);
    hipFree(device_img.grads);
    hipFree(&device_img.width);
    hipFree(&device_img.height);
}
