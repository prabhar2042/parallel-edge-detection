#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "canny.h"

#define THREADS_PER_BLOCK 16

__constant__ float blur_kernel[3][3] = {{0.0625, 0.125, 0.0625},
                                        {0.125, 0.25, 0.125},
                                        {0.0625, 0.125, 0.0625}};

__global__ void rgb_to_gray_kernel(Pixels *pixels, int height, int width)
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= height || col >= width)
    {
        return;
    }

    double grayValue = 0.299 * pixels[row * width + col].rgb.r +
                       0.587 * pixels[row * width + col].rgb.g +
                       0.114 * pixels[row * width + col].rgb.b;

    // pixels[row * width + col].gray.value = static_cast<unsigned char>(grayValue);
    pixels[row * width + col].gray.value = 0;
}

__global__ void gaussianblur_kernel(Image device_img, Image device_result)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row > (device_img.height - 1) || col > (device_img.width - 1) || row < 1 || col < 1)
    {
        return;
    }

    float sum = 0.0;
    for (int i = -1; i <= 1; ++i)
    {
        for (int j = -1; j <= 1; ++j)
        {
            sum += blur_kernel[i + 1][j + 1] * device_img.pixels[row + i][col + j].gray.value;
        }
    }
    // Store the blurred pixel in the new image
    device_result.pixels[row][col].gray.value = static_cast<unsigned char>(sum);
}
/*
__global__ void sobel_kernel(Image *img)
{
}

__global__ void nms_kernel(Image *img)
{
}

__global__ void threshholding_kernel(Image *img)
{
}

__global__ void edge_tracking_kernel(Image *img){}

*/

void canny(exec_time &time, char *read_file, char *write_file)
{
    // allocate memory for host image on stack
    Image host_img;

    printf("about to read ppm\n");

    // read image
    read_PPM(host_img, read_file);

    printf("read done \n");

    // flatten Pixel array
    Pixel host_flat_pixels[host_img.height * host_img.width];
    flatten_array(host_img, host_flat_pixels);

    // allocate memory for image on GPU
    Pixel *device_pixels;
    hipMalloc((void **)&device_pixels, host_img.width * host_img.height * sizeof(Pixel));
    printf("malloc 1 done  \n");

    // copy image to GPU
    hipMemcpy(device_pixels, host_flat_pixels, host_img.width * host_img.height * sizeof(Pixel), hipMemcpyHostToDevice);

    printf("memcpy done  \n");

    // Define block and grid dimensions
    dim3 blockDim(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 gridDim((host_img.width + blockDim.x - 1) / blockDim.x,
                 (host_img.height + blockDim.y - 1) / blockDim.y);

    double start, end;

    // start image detection on GPU

    start = clock();
    rgb_to_gray_kernel<<<blockDim, gridDim>>>(device_pixels, host_img.height, host_img.width); // 1. convert image to grayscale
    hipDeviceSynchronize();
    end = clock();
    time.rgb_to_gray = ((double)end - (double)start) / CLOCKS_PER_SEC;
    time.total += time.rgb_to_gray;

    /*
        start = clock();
        gaussianblur_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result); // 2. Gaussian Blur
        hipDeviceSynchronize();
        end = clock();
        time.gaussian_blur = ((double)end - (double)start) / CLOCKS_PER_SEC;
        time.total += time.gaussian_blur;

        start = clock();
        sobel_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result); // 3. Determine intensity gradient
        hipDeviceSynchronize();
        end = clock();
        time.sobel = ((double)end - (double)start) / CLOCKS_PER_SEC;
        time.total += time.sobel;

        start = clock();
        nms_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result); // 4. Non Maximum seperation
        hipDeviceSynchronize();
        end = clock();
        time.nms = ((double)end - (double)start) / CLOCKS_PER_SEC;
        time.total += time.nms;

        start = clock();
        threshholding_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result); // 5. Double Thresholding
        hipDeviceSynchronize();
        end = clock();
        time.double_thres = ((double)end - (double)start) / CLOCKS_PER_SEC;
        time.total += time.double_thres;

        start = clock();
        edge_tracking_kernel<<<blocks, threadsPerBlock>>>(device_img, device_result); // 6. Edge Tracking
        hipDeviceSynchronize();
        end = clock();
        time.edge_track = ((double)end - (double)start) / CLOCKS_PER_SEC;
        time.total += time.edge_track;

        */

    // Copy image data from device to host

    printf("I am here\n");
    hipMemcpy(host_flat_pixels, device_pixels, host_img.width * host_img.height * sizeof(Pixel), hipMemcpyDeviceToHost);

    unflatten_pixel_array(host_img, host_flat_pixels);

    // write image
    write_PPM(host_img, write_file);

    // free device memory space
    hipFree(device_pixels);
}
